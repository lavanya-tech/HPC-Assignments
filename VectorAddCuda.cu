//download cuda toolkit from nvidia website by selecting version and os
//nvcc vector_add.cu -o vector_add
//./vector_add
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void addVectors(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1000000;
    float *h_a, *h_b, *h_c; // host vectors
    float *d_a, *d_b, *d_c; // device vectors

    size_t bytes = n * sizeof(float);

    // Allocate memory on the host
    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_c = (float *)malloc(bytes);

    // Initialize the input vectors
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate memory on the device
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy input vectors from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy the result from device to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < n; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Error: mismatch at index %d\n", i);
            break;
        }
    }

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
